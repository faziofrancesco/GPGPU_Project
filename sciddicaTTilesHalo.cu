#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "util.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
using namespace std;

// ----------------------------------------------------------------------------
// I/O parameters used to index argv[]
// ----------------------------------------------------------------------------
#define HEADER_PATH_ID 1
#define DEM_PATH_ID 2
#define SOURCE_PATH_ID 3
#define OUTPUT_PATH_ID 4
#define STEPS_ID 5
// ----------------------------------------------------------------------------
// Simulation parameters
// ----------------------------------------------------------------------------
#define P_R 0.5
#define P_EPSILON 0.001
#define ADJACENT_CELLS 4
#define STRLEN 256

#define MASK_SIZE 3
#define TILE_SIZE_O 4
const int BLOCK_WIDTH = TILE_SIZE_O+MASK_SIZE-1; //TILE_SIZE

// ----------------------------------------------------------------------------
// Read/Write access macros linearizing single/multy layer buffer 2D indices
// ----------------------------------------------------------------------------
#define SET(M, columns, i, j, value) ((M)[(((i) * (columns)) + (j))] = (value))
#define GET(M, columns, i, j) (M[(((i) * (columns)) + (j))])
#define BUF_SET(M, rs, columns, n, i, j, value) ( (M)[( ((n)*(rs)*(columns)) + ((i)*(columns)) + (j) )] = (value) )
#define BUF_GET(M, rs, columns, n, i, j) ( M[( ((n)*(rs)*(columns)) + ((i)*(columns)) + (j) )] )

// ----------------------------------------------------------------------------
// I/O functions
// ----------------------------------------------------------------------------
void readHeaderInfo(char* path, int &nrs, int &ncs, /*double &xllcorner, double &yllcorner, double &cellsize,*/ double &nodata)
{
  FILE* f;
  
  if ( (f = fopen(path,"r") ) == 0){
    printf("%s configuration header file not found\n", path);
    exit(0);
  }

  //Reading the header
  char str[STRLEN];
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); ncs = atoi(str);      //ncs
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); nrs = atoi(str);      //nrs
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); //xllcorner = atof(str);  //xllcorner
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); //yllcorner = atof(str);  //yllcorner
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); //cellsize = atof(str);   //cellsize
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); nodata = atof(str);     //NODATA_value 
}

bool loadGrid2D(double *M, int rs, int columns, char *path)
{
  FILE *f = fopen(path, "r");

  if (!f) {
    printf("%s grid file not found\n", path);
    exit(0);
  }

  char str[STRLEN];
  for (int i = 0; i < rs; i++)
    for (int j = 0; j < columns; j++)
    {
      fscanf(f, "%s", str);
      SET(M, columns, i, j, atof(str));
    }

  fclose(f);

  return true;
}

bool saveGrid2Dr(double *M, int rs, int columns, char *path)
{
  FILE *f;
  f = fopen(path, "w");

  if (!f)
    return false;

  char str[STRLEN];
  for (int i = 0; i < rs; i++)
  {
    for (int j = 0; j < columns; j++)
    {
      sprintf(str, "%f ", GET(M, columns, i, j));
      fprintf(f, "%s ", str);
    }
    fprintf(f, "\n");
  }

  fclose(f);

  return true;
}

// ----------------------------------------------------------------------------
// init kernel, called once before the simulation loop
// ----------------------------------------------------------------------------
__global__ void sciddicaTSimulationInit_Kernel(int r, int c, double* Sz, double* Sh, int i_s, int i_e, int j_s, int j_e)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x; //righe
  int j = blockIdx.y * blockDim.y + threadIdx.y; //colonne

  if(i < i_s || i >= i_e)
    return;
  if(j < j_s || j >= j_e)
    return;
 
  double z, h;
  h = GET(Sh, c, i, j);

  if (h > 0.0)
  {
    z = GET(Sz, c, i, j);
    SET(Sz, c, i, j, z - h);
  }
  

}

// ----------------------------------------------------------------------------
// computing kernels, aka elementary processes in the XCA terminology
// ----------------------------------------------------------------------------

__global__ void sciddicaTResetFlows_Kernel(int r, int c, double nodata, double* Sf, int i_s, int i_e, int j_s, int j_e)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if(i < i_s || i >= i_e)
    return;
  if(j < j_s || j >= j_e)
    return;

    

  BUF_SET(Sf, r, c, 0, i, j, 0.0);
  BUF_SET(Sf, r, c, 1, i, j, 0.0);
  BUF_SET(Sf, r, c, 2, i, j, 0.0);
  BUF_SET(Sf, r, c, 3, i, j, 0.0);
}

__global__ void sciddicaTFlowsComputation_Kernel(int r, int c, double nodata, int* Xi, int* Xj, double *Sz, double *Sh, double *Sf, double p_r, double p_epsilon, int i_s, int i_e, int j_s, int j_e)
{
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int row_o = blockIdx.x*TILE_SIZE_O + tx; //+ i_s;
  int row_i = row_o - i_s;
  int col_o = blockIdx.y*TILE_SIZE_O + ty;// + j_s;
  int col_i = col_o - j_s;

  __shared__ double Sz_shared[BLOCK_WIDTH][BLOCK_WIDTH];
  __shared__ double Sh_shared[BLOCK_WIDTH][BLOCK_WIDTH];

  // Tutti i thread partecipano al caricamento delle tessere di input nella  memoria condivisa. Ogni thread controlla se gli indici y e 
//x dei suoi  elementi della piastrella di input rientrano nell'intervallo valido dell'input. 
// In caso contrario, l'elemento di input che sta tentando di caricare è effettivamente
// una ghost cells e un valore 0.0 viene  inserito  nella memoria condivisa.

  if((row_i >= 0) && (row_i < r)  && (col_i >= 0) && (col_i < c))
  {
    Sz_shared[tx][ty] = GET(Sz, c, row_i, col_i);
    Sh_shared[tx][ty] = GET(Sh, c, row_i, col_i);
  }
  else
  {
    Sz_shared[tx][ty] = 0.0f;
    Sh_shared[tx][ty] = 0.0f;
  }

  __syncthreads();

  bool eliminated_cells[5] = {false, false, false, false, false};
  bool again;
  double cells_count;
  double average;
  double m;
  double u[5];
  int n;
  double z, h;
  
  if(tx == BLOCK_WIDTH-1 || ty == BLOCK_WIDTH-1)
    return;

  if((blockIdx.x == 0) || (blockIdx.x == blockDim.x-1))
  {
    if(tx < 2 || tx >= i_e-1)
      return;
  }
  else
  {
    if(tx < 1 || tx >= i_e)
      return;
  }

  if((blockIdx.y == 0) || (blockIdx.y == blockDim.y-1))
  {
    if(ty < 2 || ty >= j_e-1)
      return;
  }
  else
  {
    if(ty < 1 || ty >= j_e)
      return;
  }


  m = Sh_shared[tx][ty] - p_epsilon;
  u[0] = Sz_shared[tx][ty] + p_epsilon;
  z = Sz_shared[tx - 1][ty];
  h = Sh_shared[tx - 1][ty];
  u[1] = z + h;                                         
  z = Sz_shared[tx + Xi[2]][ty + Xj[2]];
  h = Sh_shared[tx + Xi[2]][ty + Xj[2]];
  u[2] = z + h;                                         
  z = Sz_shared[tx][ty + 1];
  h = Sh_shared[tx][ty + 1];
  u[3] = z + h;                                         
  z = Sz_shared[tx + 1][ty];
  h = Sh_shared[tx + 1][ty];
  u[4] = z + h;

  do
  {
    again = false;
    average = m;
    cells_count = 0;

    for (n = 0; n < 5; n++)
      if (!eliminated_cells[n])
      {
        average += u[n];
        cells_count++;
      }

    if (cells_count != 0)
      average /= cells_count;

    for (n = 0; n < 5; n++)
    {
      if ((average <= u[n]) && (!eliminated_cells[n]))
      {
        eliminated_cells[n] = true;
        again = true;
      }
    }
  } while (again);

  if (!eliminated_cells[1]) BUF_SET(Sf, r, c, 0, row_i, col_i, (average - u[1]) * p_r);
  if (!eliminated_cells[2]) BUF_SET(Sf, r, c, 1, row_i, col_i, (average - u[2]) * p_r);
  if (!eliminated_cells[3]) BUF_SET(Sf, r, c, 2, row_i, col_i, (average - u[3]) * p_r);
  if (!eliminated_cells[4]) BUF_SET(Sf, r, c, 3, row_i, col_i, (average - u[4]) * p_r);
}

__global__ void sciddicaTWidthUpdate_Kernel(int r, int c, double nodata, int* Xi, int* Xj, double *Sz, double *Sh, double *Sf, int i_s, int i_e, int j_s, int j_e)
{
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int row_o = blockIdx.x*TILE_SIZE_O + tx;
  int row_i = row_o - i_s;
  int col_o = blockIdx.y*TILE_SIZE_O + ty;
  int col_i = col_o - j_s;

  __shared__ double shared[BLOCK_WIDTH][BLOCK_WIDTH][4];
 

  if((row_i >= 0) && (col_i >= 0) && (row_i < r) && (col_i < c))
  {
    shared[tx][ty][0] = BUF_GET(Sf, r, c, 0, row_i, col_i);
    shared[tx][ty][1] = BUF_GET(Sf, r, c, 1, row_i, col_i);
    shared[tx][ty][2] = BUF_GET(Sf, r, c, 2, row_i, col_i);
    shared[tx][ty][3] = BUF_GET(Sf, r, c, 3, row_i, col_i);
  }
  else
  {
    shared[tx][ty][0] = 0.0f;
    shared[tx][ty][1] = 0.0f;
    shared[tx][ty][2] = 0.0f;
    shared[tx][ty][3] = 0.0f;
  }

  __syncthreads();

  if(tx == BLOCK_WIDTH-1 || ty == BLOCK_WIDTH-1)
    return;

  if((blockIdx.x == 0) || (blockIdx.x == blockDim.x-1))
  {
    if(tx < 2 || tx >= i_e-1)
      return;
  }
  else
  {
    if(tx < 1 || tx >= i_e)
      return;
  }

  if((blockIdx.y == 0) || (blockIdx.y == blockDim.y-1))
  {
    if(ty < 2 || ty >= j_e-1)
      return;
  }
  else
  {
    if(ty < 1 || ty >= j_e-1)
      return;
  }

  double h_next;

  if(row_i == r-1 || col_i == c-1)
    return;

  if((row_i >= 0) && (row_i < r)  && (col_i >= 0) && (col_i < c))
  {
    h_next = GET(Sh, c, row_i, col_i);
  }
  else
    return;

  h_next += shared[tx + Xi[1]][ty + Xj[1]][3] - shared[tx][ty][0];
  h_next += shared[tx + Xi[2]][ty + Xj[2]][2] - shared[tx][ty][1];
  h_next += shared[tx + Xi[3]][ty + Xj[3]][1] - shared[tx][ty][2];
  h_next += shared[tx + Xi[4]][ty + Xj[4]][0] - shared[tx][ty][3];
  
  SET(Sh, c, row_i, col_i, h_next );
}

// ----------------------------------------------------------------------------
// Function main()
// ----------------------------------------------------------------------------
int main(int argc, char **argv)
{
  int rs, cs;
  double nodata;
  readHeaderInfo(argv[HEADER_PATH_ID], rs, cs, nodata);

  int r = rs;                  // r: grid rs
  int c = cs;                  // c: grid columns
  int i_s = 1, i_e = r-1;  // [i_s,i_e[: kernels application range along the rs
  int j_s = 1, j_e = c-1;  // [i_s,i_e[: kernels application range along the rs
  double *Sz;                    // Sz: substate (grid) containing the cells' altitude a.s.l.
  double *Sh;                    // Sh: substate (grid) containing the cells' flow thickness
  double *Sf;                    // Sf: 4 substates containing the flows towards the 4 neighs
  int *Xi;// Xj: von Neuman neighborhood row coordinates (see below)
  int *Xj;// Xj: von Neuman neighborhood col coordinates (see below)
  double p_r = P_R;              // p_r: minimization algorithm outflows dumping factor
  double p_epsilon = P_EPSILON;  // p_epsilon: frictional parameter threshold
  int steps = atoi(argv[STEPS_ID]); //steps: simulation steps

  // The adopted von Neuman neighborhood
  // Format: flow_index:cell_label:(row_index,col_index)
  //
  //   cell_label in [0,1,2,3,4]: label assigned to each cell in the neighborhood
  //   flow_index in   [0,1,2,3]: outgoing flow indices in Sf from cell 0 to the others
  //       (row_index,col_index): 2D relative indices of the cells
  //
  //               |0:1:(-1, 0)|
  //   |1:2:( 0,-1)| :0:( 0, 0)|2:3:( 0, 1)|
  //               |3:4:( 1, 0)|
  //
  //


  hipMallocManaged(&Xi, sizeof(int) * 5);
  hipMallocManaged(&Xj, sizeof(int) * 5);

  Xi[0] = 0;
  Xi[1] = -1;
  Xi[2] = 0;
  Xi[3] = 0;
  Xi[4] = 1;

  Xj[0] = 0;
  Xj[1] = 0;
  Xj[2] = -1;
  Xj[3] = 1;
  Xj[4] = 0;
  

  hipMallocManaged(&Sz, sizeof(double) * r * c);
  hipMallocManaged(&Sh, sizeof(double) * r * c);
  hipMallocManaged(&Sf, sizeof(double) * ADJACENT_CELLS * r * c);
  

 

  loadGrid2D(Sz, r, c, argv[DEM_PATH_ID]);   // Load Sz from file


  loadGrid2D(Sh, r, c, argv[SOURCE_PATH_ID]);// Load Sh from file

  


  dim3 dimGrid((r-1)/TILE_SIZE_O  , (c-1)/TILE_SIZE_O , 1);
  dim3 dimBlock(BLOCK_WIDTH,BLOCK_WIDTH,1);


  // Apply the init kernel (elementary process) to the whole domain grid (cellular space)
  
  sciddicaTSimulationInit_Kernel<<<dimGrid, dimBlock>>>(r, c, Sz, Sh, i_s, i_e, j_s, j_e);
  hipDeviceSynchronize();
  
  util::Timer cl_timer;
  // simulation loop
  for (int s = 0; s < steps; ++s)
  {
    // Apply the resetFlow kernel to the whole domain
    sciddicaTResetFlows_Kernel<<<dimGrid, dimBlock>>>(r, c, nodata, Sf, i_s, i_e, j_s, j_e);
    hipDeviceSynchronize();

    // Apply the FlowComputation kernel to the whole domain
    
    sciddicaTFlowsComputation_Kernel<<<dimGrid, dimBlock>>>(r, c, nodata, Xi, Xj, Sz, Sh, Sf, p_r, p_epsilon, i_s, i_e, j_s, j_e);
    hipDeviceSynchronize();

    // Apply the WidthUpdate mass balance kernel to the whole domain
    sciddicaTWidthUpdate_Kernel<<<dimGrid, dimBlock>>>(r, c, nodata, Xi, Xj, Sz, Sh, Sf, i_s, i_e, j_s, j_e);
    hipDeviceSynchronize();

  }
  double cl_time = static_cast<double>(cl_timer.getTimeMilliseconds()) / 1000.0;
  printf("Elapsed time: %lf [s]\n", cl_time);


  saveGrid2Dr(Sh, r, c, argv[OUTPUT_PATH_ID]);// Save Sh to file

  printf("Releasing memory...\n");
  
  hipFree(Sz);
  hipFree(Sh);
  hipFree(Sf);
  
  hipFree(Xi);
  hipFree(Xj);

  return 0;
}
