#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "util.hpp"
#include <hip/hip_runtime.h>

using namespace std;

// ----------------------------------------------------------------------------
// I/O parameters used to index argv[]
// ----------------------------------------------------------------------------
#define HEADER_PATH_ID 1
#define DEM_PATH_ID 2
#define SOURCE_PATH_ID 3
#define OUTPUT_PATH_ID 4
#define STEPS_ID 5
// ----------------------------------------------------------------------------
// Simulation parameters
// ----------------------------------------------------------------------------
#define P_R 0.5
#define P_EPSILON 0.001
#define ADJACENT_CELLS 4
#define STRLEN 256

#define TILE_SIZE_O 8

// ----------------------------------------------------------------------------
// Read/Write access macros linearizing single/multy layer buffer 2D indices
// ----------------------------------------------------------------------------
#define SET(M, columns, i, j, value) ((M)[(((i) * (columns)) + (j))] = (value))
#define GET(M, columns, i, j) (M[(((i) * (columns)) + (j))])
#define BUF_SET(M, rs, columns, n, i, j, value) ( (M)[( ((n)*(rs)*(columns)) + ((i)*(columns)) + (j) )] = (value) )
#define BUF_GET(M, rs, columns, n, i, j) ( M[( ((n)*(rs)*(columns)) + ((i)*(columns)) + (j) )] )

// ----------------------------------------------------------------------------
// I/O functions
// ----------------------------------------------------------------------------
void readHeaderInfo(char* path, int &nrs, int &ncs, /*double &xllcorner, double &yllcorner, double &cellsize,*/ double &nodata)
{
  FILE* f;
  
  if ( (f = fopen(path,"r") ) == 0){
    printf("%s configuration header file not found\n", path);
    exit(0);
  }

  //Reading the header
  char str[STRLEN];
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); ncs = atoi(str);      //ncs
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); nrs = atoi(str);      //nrs
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); //xllcorner = atof(str);  //xllcorner
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); //yllcorner = atof(str);  //yllcorner
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); //cellsize = atof(str);   //cellsize
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); nodata = atof(str);     //NODATA_value 
}

bool loadGrid2D(double *M, int rs, int columns, char *path)
{
  FILE *f = fopen(path, "r");

  if (!f) {
    printf("%s grid file not found\n", path);
    exit(0);
  }

  char str[STRLEN];
  for (int i = 0; i < rs; i++)
    for (int j = 0; j < columns; j++)
    {
      fscanf(f, "%s", str);
      SET(M, columns, i, j, atof(str));
    }

  fclose(f);

  return true;
}

bool saveGrid2Dr(double *M, int rs, int columns, char *path)
{
  FILE *f;
  f = fopen(path, "w");

  if (!f)
    return false;

  char str[STRLEN];
  for (int i = 0; i < rs; i++)
  {
    for (int j = 0; j < columns; j++)
    {
      sprintf(str, "%f ", GET(M, columns, i, j));
      fprintf(f, "%s ", str);
    }
    fprintf(f, "\n");
  }

  fclose(f);

  return true;
}

// ----------------------------------------------------------------------------
// init kernel, called once before the simulation loop
// ----------------------------------------------------------------------------
__global__ void sciddicaTSimulationInit_Kernel(int r, int c, double* Sz, double* Sh, int i_s, int i_e, int j_s, int j_e)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x; 
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if(i < i_s || i >= i_e)
    return;
  if(j < j_s || j >= j_e)
    return;
 
  double z, h;
  h = GET(Sh, c, i, j);

  if (h > 0.0)
  {
    z = GET(Sz, c, i, j);
    SET(Sz, c, i, j, z - h);
  }
  

}

// ----------------------------------------------------------------------------
// computing kernels, aka elementary processes in the XCA terminology
// ----------------------------------------------------------------------------

__global__ void sciddicaTResetFlows_Kernel(int r, int c, double nodata, double* Sf, int i_s, int i_e, int j_s, int j_e)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if(i < i_s || i >= i_e)
    return;
  if(j < j_s || j >= j_e)
    return;

  BUF_SET(Sf, r, c, 0, i, j, 0.0);
  BUF_SET(Sf, r, c, 1, i, j, 0.0);
  BUF_SET(Sf, r, c, 2, i, j, 0.0);
  BUF_SET(Sf, r, c, 3, i, j, 0.0);
}

__global__ void sciddicaTFlowsComputation_Kernel(int r, int c, double nodata, int* Xi, int* Xj, double *Sz, double *Sh, double *Sf, double p_r, double p_epsilon, int i_s, int i_e, int j_s, int j_e)
{
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int i = blockIdx.x*TILE_SIZE_O + tx;
  int j = blockIdx.y*TILE_SIZE_O + ty;

  __shared__ double Sz_shared[TILE_SIZE_O][TILE_SIZE_O];
  __shared__ double Sh_shared[TILE_SIZE_O][TILE_SIZE_O];

  Sz_shared[tx][ty] = GET(Sz, c, i, j);
  Sh_shared[tx][ty] = GET(Sh, c, i, j);

  __syncthreads();

  if(i < i_s || i >= i_e)
    return;
  if(j < j_s || j >= j_e)
    return;

  bool eliminated_cells[5] = {false, false, false, false, false};
  bool again;
  double cells_count;
  double average;
  double m;
  double u[5];
  int n;
  double z, h;


  m = Sh_shared[tx][ty] - p_epsilon;
  u[0] = Sz_shared[tx][ty] + p_epsilon;

  for(int k = 1; k < 5; k++)
  {
    if(tx + Xi[k] < 0 || tx + Xi[k] >= TILE_SIZE_O || ty + Xj[k] < 0 || ty + Xj[k] >= TILE_SIZE_O)
    {
      z = GET(Sz, c, i + Xi[k], j + Xj[k]);
      h = GET(Sh, c, i + Xi[k], j + Xj[k]);
    }
    else
    {
      z = Sz_shared[tx + Xi[k]][ty + Xj[k]];
      h = Sh_shared[tx + Xi[k]][ty + Xj[k]];
    }  
    u[k] = z + h;
  }

  do
  {
    again = false;
    average = m;
    cells_count = 0;

    for (n = 0; n < 5; n++)
      if (!eliminated_cells[n])
      {
        average += u[n];
        cells_count++;
      }

    if (cells_count != 0)
      average /= cells_count;

    for (n = 0; n < 5; n++)
    {
      if ((average <= u[n]) && (!eliminated_cells[n]))
      {
        eliminated_cells[n] = true;
        again = true;
      }
    }
  } while (again);

  if (!eliminated_cells[1]) BUF_SET(Sf, r, c, 0, i, j, (average - u[1]) * p_r);
  if (!eliminated_cells[2]) BUF_SET(Sf, r, c, 1, i, j, (average - u[2]) * p_r);
  if (!eliminated_cells[3]) BUF_SET(Sf, r, c, 2, i, j, (average - u[3]) * p_r);
  if (!eliminated_cells[4]) BUF_SET(Sf, r, c, 3, i, j, (average - u[4]) * p_r);
}

__global__ void sciddicaTWidthUpdate_Kernel(int r, int c, double nodata, int* Xi, int* Xj, double *Sz, double *Sh, double *Sf, int i_s, int i_e, int j_s, int j_e)
{
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  __shared__ double shared[TILE_SIZE_O][TILE_SIZE_O][4];
 
  shared[tx][ty][0] = BUF_GET(Sf, r, c, 0, i, j);
  shared[tx][ty][1] = BUF_GET(Sf, r, c, 1, i, j);
  shared[tx][ty][2] = BUF_GET(Sf, r, c, 2, i, j);
  shared[tx][ty][3] = BUF_GET(Sf, r, c, 3, i, j);

  __syncthreads();

  if(i < i_s || i >= i_e)
    return;
  if(j < j_s || j >= j_e)
    return;

  double h_next;

  h_next = GET(Sh, c, i, j);

  for(int k = 1, k_inv = 3; k_inv >= 0; k++, k_inv = k_inv-1)
  {
    if(tx + Xi[k] < 0 || tx + Xi[k] >= TILE_SIZE_O || ty + Xj[k] < 0 || ty + Xj[k] >= TILE_SIZE_O)
    {
      h_next += BUF_GET(Sf, r, c, k_inv, i+Xi[k], j+Xj[k]) - shared[tx][ty][k-1];
    }
    else
    {
      h_next += shared[tx + Xi[k]][ty + Xj[k]][k_inv] - shared[tx][ty][k-1];
    }
  }

  SET(Sh, c, i, j, h_next );
}

// ----------------------------------------------------------------------------
// Function main()
// ----------------------------------------------------------------------------
int main(int argc, char **argv)
{
  int rs, cs;
  double nodata;
  readHeaderInfo(argv[HEADER_PATH_ID], rs, cs, nodata);

  int r = rs;                  // r: grid rs
  int c = cs;                  // c: grid columns
  int i_s = 1, i_e = r-1;  // [i_s,i_e[: kernels application range along the rs
  int j_s = 1, j_e = c-1;  // [i_s,i_e[: kernels application range along the rs
  double *Sz;                    // Sz: substate (grid) containing the cells' altitude a.s.l.
  double *Sh;                    // Sh: substate (grid) containing the cells' flow thickness
  double *Sf;                    // Sf: 4 substates containing the flows towards the 4 neighs
  int *Xi;// Xj: von Neuman neighborhood row coordinates (see below)
  int *Xj;// Xj: von Neuman neighborhood col coordinates (see below)
  double p_r = P_R;              // p_r: minimization algorithm outflows dumping factor
  double p_epsilon = P_EPSILON;  // p_epsilon: frictional parameter threshold
  int steps = atoi(argv[STEPS_ID]); //steps: simulation steps

  // The adopted von Neuman neighborhood
  // Format: flow_index:cell_label:(row_index,col_index)
  //
  //   cell_label in [0,1,2,3,4]: label assigned to each cell in the neighborhood
  //   flow_index in   [0,1,2,3]: outgoing flow indices in Sf from cell 0 to the others
  //       (row_index,col_index): 2D relative indices of the cells
  //
  //               |0:1:(-1, 0)|
  //   |1:2:( 0,-1)| :0:( 0, 0)|2:3:( 0, 1)|
  //               |3:4:( 1, 0)|
  //
  //

  hipMallocManaged(&Xi, sizeof(int) * 5);
  hipMallocManaged(&Xj, sizeof(int) * 5);

  Xi[0] = 0;
  Xi[1] = -1;
  Xi[2] = 0;
  Xi[3] = 0;
  Xi[4] = 1;

  Xj[0] = 0;
  Xj[1] = 0;
  Xj[2] = -1;
  Xj[3] = 1;
  Xj[4] = 0;
  

  hipMallocManaged(&Sz, sizeof(double) * r * c);
  hipMallocManaged(&Sh, sizeof(double) * r * c);
  hipMallocManaged(&Sf, sizeof(double) * ADJACENT_CELLS * r * c);
  
  loadGrid2D(Sz, r, c, argv[DEM_PATH_ID]);   // Load Sz from file


  loadGrid2D(Sh, r, c, argv[SOURCE_PATH_ID]);// Load Sh from file


  

  dim3 dimGrid((r-1)/TILE_SIZE_O , (c-1)/TILE_SIZE_O , 1);
  dim3 dimBlock(TILE_SIZE_O,TILE_SIZE_O,1);

  // Apply the init kernel (elementary process) to the whole domain grid (cellular space)
  
  sciddicaTSimulationInit_Kernel<<<dimGrid, dimBlock>>>(r, c, Sz, Sh, i_s, i_e, j_s, j_e);
  hipDeviceSynchronize();
  

  util::Timer cl_timer;
  // simulation loop
  for (int s = 0; s < steps; ++s)
  {
    // Apply the resetFlow kernel to the whole domain
    sciddicaTResetFlows_Kernel<<<dimGrid, dimBlock>>>(r, c, nodata, Sf, i_s, i_e, j_s, j_e);
    hipDeviceSynchronize();

    // Apply the FlowComputation kernel to the whole domain
    
    sciddicaTFlowsComputation_Kernel<<<dimGrid, dimBlock>>>(r, c, nodata, Xi, Xj, Sz, Sh, Sf, p_r, p_epsilon, i_s, i_e, j_s, j_e);
    hipDeviceSynchronize();

    // Apply the WidthUpdate mass balance kernel to the whole domain
    sciddicaTWidthUpdate_Kernel<<<dimGrid, dimBlock>>>(r, c, nodata, Xi, Xj, Sz, Sh, Sf, i_s, i_e, j_s, j_e);
    hipDeviceSynchronize();

  }
  double cl_time = static_cast<double>(cl_timer.getTimeMilliseconds()) / 1000.0;
  printf("Elapsed time: %lf [s]\n", cl_time);


  

  saveGrid2Dr(Sh, r, c, argv[OUTPUT_PATH_ID]);// Save Sh to file

  printf("Releasing memory...\n");
  
  hipFree(Sz);
  hipFree(Sh);
  hipFree(Sf);
  
  hipFree(Xi);
  hipFree(Xj);

  return 0;
}
